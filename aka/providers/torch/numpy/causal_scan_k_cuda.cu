#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace { namespace device {
    template <typename scalar_t> __global__ void causalScan4d_Forward(
        int b, int l, int h, int d,
        scalar_t * Z,
        scalar_t * A,
        scalar_t * B,
        scalar_t * O
    )
    {
        int idx_A = ((blockIdx.x % b) * l * h + blockIdx.y) * d + blockIdx.z % d;
        int idx_B = (blockIdx.x * l * gridDim.y + blockIdx.y) * gridDim.z + blockIdx.z;
        int idx_Z = (blockIdx.x * gridDim.y + blockIdx.y) * gridDim.z + blockIdx.z;
        int step_A = h * d;
        int step_B = gridDim.y * gridDim.z;
        scalar_t zh = Z[idx_Z];
        while(l-->0) {
            zh = A[idx_A] * zh + B[idx_B];
            O[idx_B] = zh;
            idx_A += step_A;
            idx_B += step_B;
        }
    }

    template <typename scalar_t> __global__ void causalScan4d_Backward(
        int b, int l, int h, int d,
        scalar_t * gradZ,
        scalar_t * gradA,
        scalar_t * gradB,
        scalar_t * gradO,
        scalar_t * Z,
        scalar_t * A,
        scalar_t * O
    )
    {
        int idx_A = ((blockIdx.x % b) * l * h + blockIdx.y) * d + blockIdx.z % d;
        int idx_B = (blockIdx.x * l * gridDim.y + blockIdx.y) * gridDim.z + blockIdx.z;
        int idx_Z = (blockIdx.x * gridDim.y + blockIdx.y) * gridDim.z + blockIdx.z;
        int step_A = h * d;
        int step_B = gridDim.y * gridDim.z;
        idx_A += step_A * (l - 1);
        idx_B += step_B * (l - 1);
        scalar_t grad = 0.0;
        while(l-->1) {
            grad += gradO[idx_B];
            gradB[idx_B] = grad;
            atomicAdd(gradA + idx_A, O[idx_B - step_B] * grad);
            grad *= A[idx_A];
            idx_A -= step_A;
            idx_B -= step_B;
        }
        grad += gradO[idx_B];
        gradB[idx_B] = grad;
        atomicAdd(gradA + idx_A, Z[idx_Z] * grad);
        gradZ[idx_Z] = A[idx_A] * grad;
    }
}}

#include <torch/extension.h>
#include <vector>

torch::Tensor causalScan4d_cpu_Forward(torch::Tensor Z, torch::Tensor A, torch::Tensor B);
torch::Tensor causalScan4d_cuda_Forward(torch::Tensor Z, torch::Tensor A, torch::Tensor B) {
    if(!A.is_cuda()) {
        return causalScan4d_cpu_Forward(Z,A,B);
    }
    auto O = torch::zeros_like(B);
    int ba = A.size(0);
    int la = A.size(1);
    int ha = A.size(2);
    int da = A.size(3);
    const dim3 blocks(O.size(0), O.size(2), O.size(3));
    AT_DISPATCH_FLOATING_TYPES(O.type(), "causalScan4d_Forward", ([&] {
        device::causalScan4d_Forward<scalar_t><<<blocks, 1>>>(
            ba, la, ha, da,
            (scalar_t*)Z.data_ptr(),
            (scalar_t*)A.data_ptr(),
            (scalar_t*)B.data_ptr(),
            (scalar_t*)O.data_ptr()
        );
    }));
    return O;
}

std::vector<torch::Tensor> causalScan4d_cpu_Backward(torch::Tensor gradO, torch::Tensor Z, torch::Tensor A, torch::Tensor O);
std::vector<torch::Tensor> causalScan4d_cuda_Backward(torch::Tensor gradO, torch::Tensor Z, torch::Tensor A, torch::Tensor O) {
    if(!A.is_cuda()) {
        return causalScan4d_cpu_Backward(gradO,Z,A,O);
    }
    auto gradA = torch::zeros_like(A);
    auto gradB = torch::zeros_like(O);
    auto gradZ = torch::zeros_like(O.slice(1, 0, 1));
    int ba = A.size(0);
    int la = A.size(1);
    int ha = A.size(2);
    int da = A.size(3);
    const dim3 blocks(O.size(0), O.size(2), O.size(3));
    AT_DISPATCH_FLOATING_TYPES(O.type(), "causalScan4d_Backward", ([&] {
        device::causalScan4d_Backward<scalar_t><<<blocks, 1>>>(
            ba, la, ha, da,
            (scalar_t*)gradZ.data_ptr(),
            (scalar_t*)gradA.data_ptr(),
            (scalar_t*)gradB.data_ptr(),
            (scalar_t*)gradO.data_ptr(),
            (scalar_t*)Z.data_ptr(),
            (scalar_t*)A.data_ptr(),
            (scalar_t*)O.data_ptr()
        );
    }));
    return {gradZ, gradA, gradB};
}
